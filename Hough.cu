#include "hip/hip_runtime.h"
#include "Hough.h"
#include <vector>
#include <string>
#include <iostream>
#include "Rivelatore.h"
#include <math.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform.h>

void VecToTrust(std::vector<float> &v)
{
  std::cout << "--Mucca" << std::endl;
  thrust::device_vector<float> D(v.begin(), v.end());
  print_vector("Test",D);
}

// sparse histogram using reduce_by_key
template <typename Vector1,
          typename Vector2,
          typename Vector3>
void sparse_histogram(Vector1& data,
                            Vector2& histogram_values,
                            Vector3& histogram_counts)
{
  typedef typename Vector1::value_type ValueType; // input value type
  typedef typename Vector3::value_type IndexType; // histogram index type

  /*
  // copy input data (could be skipped if input is allowed to be modified)
  thrust::device_vector<ValueType> data(input);
    
  // print the initial data
  print_vector("initial data", data);
  */

  // sort data to bring equal elements together
  thrust::sort(data.begin(), data.end());
  
  // print the sorted data
  //print_vector("sorted data", data);

  // number of histogram bins is equal to number of unique values (assumes data.size() > 0)
  IndexType num_bins = thrust::inner_product(data.begin(), data.end() - 1,
                                             data.begin() + 1,
                                             IndexType(1),
                                             thrust::plus<IndexType>(),
                                             thrust::not_equal_to<ValueType>());

  // resize histogram storage
  histogram_values.resize(num_bins);
  histogram_counts.resize(num_bins);
  
  // compact find the end of each bin of values
  thrust::reduce_by_key(data.begin(), data.end(),
                        thrust::constant_iterator<IndexType>(1),
                        histogram_values.begin(),
                        histogram_counts.begin());
  
  // print the sparse histogram
  //print_vector("histogram values", histogram_values);
  //print_vector("histogram counts", histogram_counts);
}


void calculateRho(std::vector<std::vector<std::vector<int>>> &values, std::vector<int> &max, std::vector<float> &yValueFloat, std::vector<float> &xValueFloat, const float thetaPrecision, const float rhoPrecision, const float ymax, bool costrain)
{
  thrust::device_vector<float> xValueFloatTrust(xValueFloat.begin(), xValueFloat.end());
  thrust::device_vector<float> yValueFloatTrust(yValueFloat.begin(), yValueFloat.end());

  //thrust::device_vector<float> cosValue(xValueFloatTrust.size()); //Creatin of vector for cos value
  //thrust::device_vector<float> sinValue(xValueFloatTrust.size()); //Creatin of vector for sin value

  //Termporari vectors to store x*cos and y*sin
  thrust::device_vector<float> xTemp(xValueFloatTrust.size());
  thrust::device_vector<float> yTemp(xValueFloatTrust.size());

  //Vector temporary containin non dicreta values of rho
  thrust::device_vector<float> rhoTemp(xValueFloatTrust.size());

  //Vector containing fila rho discrete values
  thrust::device_vector<int> rho(xValueFloatTrust.size());

  //Detector for histogram
  thrust::device_vector<int> histogram_values;
  thrust::device_vector<int> histogram_counts;

  float angle = 0;

  for (int i = 0; i < int(values.size()); i++)
  {
    //std::cout << i+1 << std::endl;
    /*
    //Creatin of vector for cos value 
    cosValue.clear();
    thrust::fill(cosValue.begin(), cosValue.end(), cos(((i+1)*thetaPrecision*M_PI)/180));
    */

    angle = ((i+1)*thetaPrecision*M_PI)/180;

    //Calculation of cos(theta)*x
    thrust::transform(xValueFloatTrust.begin(), xValueFloatTrust.end(), xTemp.begin(), floatMultiplication(cos(angle)));
    //thrust::transform(xValueFloatTrust.begin(), xValueFloatTrust.end(), cosValue.begin(), xTemp.begin(), thrust::multiplies<float>());

    /*
    //Creatin of vector for sin value
    sinValue.clear();
    //thrust::fill(sinValue.begin(), sinValue.end(), sin(((i+1)*thetaPrecision*M_PI)/180));
    */

    //Calculation of sin(theta)*y
    thrust::transform(yValueFloatTrust.begin(), yValueFloatTrust.end(), yTemp.begin(), floatMultiplication(sin(angle)));
    //thrust::transform(yValueFloatTrust.begin(), yValueFloatTrust.end(), sinValue.begin(), yTemp.begin(), thrust::multiplies<float>());

    //Calulate sum
    thrust::transform(xTemp.begin(), xTemp.end(), yTemp.begin(), rhoTemp.begin(), thrust::plus<float>());

    //Calculate rho discrete
    thrust::transform(rhoTemp.begin(), rhoTemp.end(), rho.begin(), intDivision(rhoPrecision));

    /*
    print_vector("X", xValueFloatTrust);
    print_vector("COS", xTemp);
    print_vector("Y",yValueFloatTrust);
    print_vector("SIN",yTemp);
    print_vector("RHO",rhoTemp);
    print_vector("RHOD",rho);
    */

    sparse_histogram(rho, histogram_values, histogram_counts);

    std::vector<int> histoValue(histogram_values.size());
    thrust::copy(histogram_values.begin(), histogram_values.end(), histoValue.begin());
    
    std::vector<int> histoCount(histogram_values.size());
    thrust::copy(histogram_counts.begin(), histogram_counts.end(), histoCount.begin());
    
    values.at(i).push_back(histoValue);
    values.at(i).push_back(histoCount);
    
    for (int j = 0; j < int(histogram_values.size()); j++)
    {
        //Calculate value for fit (angle, rho, significance)
        if (histogram_counts[j] > max.at(2))
        {
            if (costrain)
            {
                float y0 = (histogram_values[j]*rhoPrecision)/(sin(((i+1)*thetaPrecision*M_PI)/180));
                if (y0 > 0 && y0 <ymax)
                {
                    max.at(0) = i;
                    max.at(1) = histogram_values[j];
                    max.at(2) = histogram_counts[j];
                }
            }
            else
            {
                max.at(0) = i;
                max.at(1) = histogram_values[j];
                max.at(2) = histogram_counts[j];
            }
        }

        //Calculate max rho
        if (histogram_values[j] > max.at(3))
            max.at(3) = histogram_values[j];
    }

  }
}
